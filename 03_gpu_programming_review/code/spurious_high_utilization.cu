// The purpose of this code is to illustrate that 100% GPU utilization as
// measured using nvidia-smi does not necessarily mean that the GPU is being
// efficiently used. In this case only a single thread is used.


#include <hip/hip_runtime.h>
#include <stdio.h>

void CPUFunction() {
  printf("\"Hello world\" from the CPU.\n");
}

__global__ void GPUFunction() {
  printf("\"Hello\" from the one and only GPU thread (index %d).\n", threadIdx.x);
  while (true) {}; // infinite loop
}

int main() {
  // function to run on the cpu
  CPUFunction();

  // function to run on the gpu
  GPUFunction<<<1, 1>>>();
  
  // kernel execution is asynchronous so sync on its completion
  hipDeviceSynchronize();
}
